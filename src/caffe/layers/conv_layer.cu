#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/layers/conv_layer.hpp"
#include "caffe/adaptive_probabilistic_pruning.hpp"
#define SHOW_INTERVAL 10
#define SHOW_NUM_LAYER 5
#define LAYER_PRINTED 0

using namespace std;
namespace caffe {

template <typename Dtype>
void ConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
          
    /// ADDED BY WANGHUAN -----------------------------------
    Dtype* muweight = this->blobs_[0]->mutable_cpu_data();
    const int count = this->blobs_[0]->count();
    const int num_row = this->blobs_[0]->shape()[0];
    const int num_col = count / num_row;
    const string layer_name = this->layer_param_.name();
    const string mthd = APP<Dtype>::prune_method;
    const int L = APP<Dtype>::layer_index[layer_name];
    this->IF_restore = false;
    
    /// IF_prune
    const bool IF_want_prune  = mthd != "None" && APP<Dtype>::prune_ratio[L] > 0; // if you want to prune, you must specify a meaningful prune_method and give a positive prune_ratio
    const bool IF_been_pruned = APP<Dtype>::pruned_ratio[L] > 0; // for a pruned layer, continue to prune
    const bool IF_enough_iter = APP<Dtype>::step_ >= APP<Dtype>::prune_begin_iter+1; // for a raw layer, if iter is enough, then prune
    this->IF_prune = IF_want_prune && (IF_been_pruned || IF_enough_iter);
    
    if (this->phase_ == TRAIN) {
        if (this->IF_prune) {
            if (APP<Dtype>::IF_update_row_col) {
                // UpdateNumPrunedRow/Col
                // Note that, UpdateNumPrunedRow/Col before pruning, 
                // so that when calculating score, the zombie weights will not be counted.
                if (APP<Dtype>::prune_unit == "Col" && L != APP<Dtype>::conv_layer_cnt-1) {
                    if (APP<Dtype>::step_-1 - APP<Dtype>::iter_prune_finished[L+1] <= 1) {
                        UpdateNumPrunedRow();
                    }
                } else if (APP<Dtype>::prune_unit == "Row" && mthd != "TP_Row" && L != 0 && APP<Dtype>::pruned_rows.size()) {
                    UpdateNumPrunedCol();
                } /// Note we don't update column for TP, because their method didn't mention this.
                UpdatePrunedRatio();
            }
            
            // check if prune finished, get into here ONLY once
            if (APP<Dtype>::iter_prune_finished[L] == INT_MAX) {
                Dtype pruned_ratio;
                if (APP<Dtype>::prune_unit == "Weight")   { pruned_ratio = APP<Dtype>::pruned_ratio[L];     }
                else if (APP<Dtype>::prune_unit == "Row") { pruned_ratio = APP<Dtype>::pruned_ratio_row[L]; }
                else if (APP<Dtype>::prune_unit == "Col") { pruned_ratio = APP<Dtype>::pruned_ratio_col[L]; }
                const bool layer_finish     = pruned_ratio >= APP<Dtype>::prune_ratio[L]; /// layer pruning target achieved
                const bool net_finish_speed = APP<Dtype>::IF_speedup_achieved;   /// net pruning target of speed achieved
                const bool net_finish_param = APP<Dtype>::IF_compRatio_achieved; /// net pruning target of compression achieved
                
                if (layer_finish || net_finish_speed || net_finish_param) {
                    APP<Dtype>::iter_prune_finished[L] = APP<Dtype>::step_ - 1;
                    
                    char* mthd = new char[strlen(APP<Dtype>::prune_method.c_str()) + 1];
                    strcpy(mthd, APP<Dtype>::prune_method.c_str());
                    const string mthd_ = strtok(mthd, "_"); // mthd is like "Reg_Col", the first split is `Reg`
                    if (mthd_ == "SPP") { CleanWorkForPP(); } // last time, do some clean work
                    
                    // print to log
                    char rlayer[10];
                    char rrow[10];
                    char rcol[10];
                    sprintf(rlayer, "%6.4f", APP<Dtype>::pruned_ratio[L]);
                    sprintf(rrow,   "%6.4f", APP<Dtype>::pruned_ratio_row[L]);
                    sprintf(rcol,   "%6.4f", APP<Dtype>::pruned_ratio_col[L]);
                    cout << layer_name << " prune finished!" 
                         << "  step: " << APP<Dtype>::step_
                         << "  net speedup: " << APP<Dtype>::speedup
                         << "  net compRatio: " << APP<Dtype>::compRatio
                         << "  pruned_ratio: " << rlayer
                         << "  pruned_ratio_row: " << rrow
                         << "  pruned_ratio_col: " << rcol 
                         << "  prune_ratio: " << APP<Dtype>::prune_ratio[L] << endl;
                    IF_alpf();
                }
            }
        }
        
        // Print and check, before update probs
        // put this outside, to print even when we do not prune
        if (L == LAYER_PRINTED && APP<Dtype>::step_ % SHOW_INTERVAL == 0 && APP<Dtype>::inner_iter == 0) {
            Print(L, 'f');
        }

        // Update masks and apply masks
        if (this->IF_prune && APP<Dtype>::iter_prune_finished[L] == INT_MAX) {
            if (mthd == "FP" && (APP<Dtype>::step_ - 1) % APP<Dtype>::prune_interval == 0) {
                FilterPrune(); 
            } else if (mthd.substr(0, 3) == "PPc" && IF_hppf()) {
                if (APP<Dtype>::prune_interval) {
                    ProbPruneCol(APP<Dtype>::prune_interval);
                } else {
                    ProbPruneCol();
                }
            } else if (mthd == "PPr" && IF_hppf()) {
                ProbPruneRow();
            } else if (mthd == "Reg_Col" || mthd == "Reg_Weight") {
                PruneMinimals();
            }
            UpdatePrunedRatio();
            if (L == APP<Dtype>::conv_layer_cnt + APP<Dtype>::fc_layer_cnt - 1) {
                APP<Dtype>::pruned_rows.clear();
            }
        }
        
        
        // Print 
        if (mthd != "None" && L < SHOW_NUM_LAYER && APP<Dtype>::inner_iter == 0) {
            cout << layer_name << "  IF_prune: " << this->IF_prune 
                 << "  pruned_ratio: " << APP<Dtype>::pruned_ratio[L];
            if (mthd == "PPr" || mthd == "FP" || mthd == "TP") {
                cout << "  pruned_ratio_col: " << APP<Dtype>::num_pruned_col[L] * 1.0 / num_col << "(" << APP<Dtype>::num_pruned_col[L] << ")"
                     << "  pruned_ratio_row: " << APP<Dtype>::num_pruned_row[L] * 1.0 / num_row << "(" << APP<Dtype>::num_pruned_row[L] << ")";
            } else {
                cout << "  pruned_ratio_row: " << APP<Dtype>::num_pruned_row[L] * 1.0 / num_row << "(" << APP<Dtype>::num_pruned_row[L] << ")"
                     << "  pruned_ratio_col: " << APP<Dtype>::num_pruned_col[L] * 1.0 / num_col << "(" << APP<Dtype>::num_pruned_col[L] << ")";
            }
            cout << "  prune_ratio: "  << APP<Dtype>::prune_ratio[L] 
                 << "  reg: " << APP<Dtype>::reg_to_distribute[L] 
                 << "/" << ceil(APP<Dtype>::prune_ratio[L] * num_col) * APP<Dtype>::target_reg << endl; 
        }
        
        
        // Weight logging
        if (APP<Dtype>::num_log) {
            const int num_log = APP<Dtype>::log_index[L].size();
            for (int k = 0; k < num_log; ++k) {
                const int index = APP<Dtype>::log_index[L][k];
                Dtype sum = 0;
                for (int i = 0; i < num_row; ++i) {
                    sum += fabs(muweight[i * num_col + index]);
                }
                sum /= num_row;
                APP<Dtype>::log_weight[L][k].push_back(sum);
            }
        }
    } else {
        if (this->IF_prune && APP<Dtype>::iter_prune_finished[L] == INT_MAX && mthd.substr(0, 2) == "PP") {
            Dtype rands[num_col];
            caffe_rng_uniform(num_col, (Dtype)0, (Dtype)1, rands);
            for (int i = 0; i < count; ++i) {
                APP<Dtype>::masks[L][i] = rands[i % num_col] < APP<Dtype>::history_prob[L][i % num_col] ? 1 : 0; /// generate masks
            }              
            for (int i = 0; i < count; ++i) { 
                this->weight_backup[i] = muweight[i]; /// backup weights
            } 
            this->IF_restore = true;
            for (int i = 0; i < count; ++i) { 
                muweight[i] *= APP<Dtype>::masks[L][i]; /// apply masks
            } 
        }
    }
  /// ------------------------------------------------------
  
    const Dtype* weight = this->blobs_[0]->gpu_data();
    for (int i = 0; i < bottom.size(); ++i) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        Dtype* top_data = top[i]->mutable_gpu_data();
        for (int n = 0; n < this->num_; ++n) {
            this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
            top_data + n * this->top_dim_);
            if (this->bias_term_) {
                const Dtype* bias = this->blobs_[1]->gpu_data();
                this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
            }
        }
    }
    /// this->bottom_dim_: bottom feature map size, input
    /// this->top_dim_: top feature map size, output
    /// this->num_: batch size
    
    /// Print feature map to check --------
    /// If row 3 and 8 are pruned in previous layer, then channel 3 and 8 will be only biases in this layer's feature map.
    /**
    if (!APP<Dtype>::IN_TEST && L == 0) {
        cout << "bottom.size(): " << bottom.size() << endl;
        for (int i = 0; i < bottom.size(); ++i) {
            const Dtype* top_data = top[i]->cpu_data();
            const int channel = top[i]->shape()[1];
            const int width   = top[i]->shape()[2];
            const int height  = top[i]->shape()[3];
            cout << "channel: " << channel << " " << width << " " <<  height << endl;
            
            vector<Dtype> sum(channel, 0);
            for (int c = 0; c < channel; ++c) {
                for (int w = 0 ; w < width; ++w) {
                    for (int h = 0; h < height; ++h) {
                        sum[c] += fabs(top_data[0 + c * width * height + w * height + h]);
                    }
                }
            }
            for (int c = 0; c < channel; ++c) {
                cout << sum[c] << "  ";
            }
            cout << endl;
        }
    }
    */
    /// -----------------------------------
    
    
    
    /// Restore weights ----------------
    if (this->IF_restore) {
        /// cout << layer_name << ": restore weights! " << endl;
        this->blobs_[0]->mutable_cpu_data();
        /// this->blobs_[0]->gpu_data(); 
        /// Interesting! If the above line is added, something like "control" seems to transfer from cpu to gpu. 
        /// Then modifying cpu weights won't affect their gpu counterparts.
        for (int i = 0; i < count; ++i) {
            muweight[i] = this->weight_backup[i];
        }
        
        /**
        /// ========================
        /// Chech restore
        cout << "weights from cpu:" << endl;
        for (int i = 0; i < 20; ++i) {
            cout << muweight[i] << " ";
        }
        cout << endl;

        Dtype weight_cpu[count];
        const Dtype* weight_gpu = this->blobs_[0]->gpu_data();
        cout << "weights copied from gpu:" << endl;
        hipMemcpy(weight_cpu, weight_gpu, sizeof(Dtype) * count, hipMemcpyDeviceToHost);
        for (int i = 0; i < 20; ++i) {
            cout << weight_cpu[i] << " ";
        }
        cout << endl;
        /// ========================
        */
    }
    /// --------------------------------
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff);
        }
        
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_);
        }
      }
    }
  }
  
/// ADDED BY WANGHUAN ------------------------------------------
    Dtype* muweight_diff = this->blobs_[0]->mutable_cpu_diff();      
    const int count   = this->blobs_[0]->count();
    const int num_row = this->blobs_[0]->shape()[0];
    const int num_col = count / num_row;
    const int L = APP<Dtype>::layer_index[this->layer_param_.name()];

    /// Diff log
    if (APP<Dtype>::num_log) {
        const int num_log = APP<Dtype>::log_index[L].size();
        for (int i = 0; i < num_log; ++i) {
            const int index = APP<Dtype>::log_index[L][i];
            Dtype sum = 0;
            for (int r = 0; r < num_row; ++r) {
                sum += fabs(muweight_diff[r * num_col + index]);
            }
            sum /= num_row;
            APP<Dtype>::log_diff[L][i].push_back(sum);
        }
    }
    
    // TaylorPrune
    if (this->IF_prune && APP<Dtype>::iter_prune_finished[L] == INT_MAX) {
        if (APP<Dtype>::prune_method == "TP" && (APP<Dtype>::step_ - 1) % APP<Dtype>::prune_interval == 0) {
            TaylorPrune(top);
        }
    }
    
    /// Print and check
    if (L == LAYER_PRINTED && APP<Dtype>::step_ % SHOW_INTERVAL == 0 && APP<Dtype>::inner_iter == 0) {
        Print(L, 'b');
    }
    
    if (APP<Dtype>::prune_method != "None" && APP<Dtype>::pruned_ratio[L] > 0) { 
        for (int j = 0; j < count; ++j) { 
            muweight_diff[j] *= APP<Dtype>::masks[L][j]; 
        }
        
        // Trying: update this to GPU code
        /* 
        caffe_gpu_mul(this->blobs_[0]->count(), 
                      this->blobs_[0]->gpu_diff(), 
                      &(APP<Dtype>::masks[L][0]), 
                      this->blobs_[0]->mutable_gpu_diff());
        
        cout << this->layer_param_.name() << " - weight_diff:" << endl;
        for (int j = 0; j < 20;  ++j) {
            cout << muweight_diff[j] << endl;
        }
        */
    }
/// ------------------------------------------------------------- 
  
  
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayer);

}  // namespace caffe
