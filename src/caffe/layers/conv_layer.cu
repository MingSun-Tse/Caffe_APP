#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/layers/conv_layer.hpp"
#include "caffe/deep_compression.hpp"
#define SHOW_INTERVAL 1

using namespace std;

namespace caffe {

template <typename Dtype>
void ConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

    /// ADDED BY WANGHUAN -----------------------------------
    Dtype* muweight = this->blobs_[0]->mutable_cpu_data();
    const int count = this->blobs_[0]->count();
    const int num_row = this->blobs_[0]->shape()[0];
    const int num_col = count / num_row;
    const string layer_name = this->layer_param_.name();
    if (this->layer_index > DeepCompression::max_layer_index) { DeepCompression::max_layer_index = this->layer_index; }
    const bool IF_mask =  DeepCompression::IN_RETRAIN 
                                 || (DeepCompression::step_ - 1) >= DeepCompression::prune_begin_iter;
    vector<Dtype> weight_backup(count, 0);
    this->IF_restore = false;
    
    /// Check -------------------------------------------
    /**
    if (!DeepCompression::IN_TEST && this->layer_index == 0) {
        for (int j = 0; j < num_col; ++j) { muweight[1  * num_col + j] = 0; }
        for (int j = 0; j < num_col; ++j) { muweight[3  * num_col + j] = 0; }
        for (int j = 0; j < num_col; ++j) { muweight[12 * num_col + j] = 0; }
        for (int j = 0; j < num_col; ++j) { muweight[16 * num_col + j] = 0; }
        for (int j = 0; j < num_col; ++j) { muweight[24 * num_col + j] = 0; }
        for (int j = 0; j < num_col; ++j) { muweight[27 * num_col + j] = 0; }
        /// for (int j = 0; j < num_col; ++j) { muweight[31 * num_col + j] = 0; }
        /// for (int j = 0; j < num_col; ++j) { muweight[37 * num_col + j] = 0; }
        /// for (int j = 0; j < num_col; ++j) { muweight[42 * num_col + j] = 0; }
    }
    */
    /// -------------------------------------------------
    
    if (this->phase_ == 0) {
        /// UpdateNumPrunedRow();
        /// UpdateNumPrunedCol();
        const Dtype pruned_ratio = 1 - (1 - this->num_pruned_column * 1.0 / num_col) * (1 - this->num_pruned_row * 1.0 / num_row);
        
        /// Print and check
        if (this->layer_index < 5 && DeepCompression::inner_iter == 0) {
            cout << layer_name << "  IF_mask: " << IF_mask << "  pruned_ratio: ";
            cout.width(3); cout << pruned_ratio << "  prune_ratio: " << this->prune_ratio << endl;
        }
                            
        /// Update masks and apply masks
        if (IF_mask && pruned_ratio < this->prune_ratio) {
            
            /// Print and check (before pruning)
            if (this->layer_index == 1 && DeepCompression::step_ % SHOW_INTERVAL == 0 && DeepCompression::inner_iter == 0) {
                /// cout.setf(std::ios::left);
                cout.width(5);  cout << "Index" << "   ";
                cout.width(18); cout << "WeightBeforeMasked" << "   ";
                cout.width(4);  cout << "Mask" << "   ";
                cout.width(4);  cout << "Prob" << endl;
                for (int i = 0; i < 20; ++i) {
                    cout.width(3);  cout << "#";
                    cout.width(2);  cout << i+1 << "   ";
                    cout.width(18); cout << muweight[i] << "   ";
                    cout.width(4);  cout << this->masks_[i] << "   ";
                    cout.width(4);  cout << DeepCompression::history_prob[this->layer_index][i] << endl;
                }
            }
            
            if (DeepCompression::prune_method == "Prune" && DeepCompression::criteria == "L2-norm") { 
                /// UpdateMasks(); 
            } else if (DeepCompression::prune_method == "FP") { 
                CHECK_GE(DeepCompression::prune_interval, 1)
                        << "Error: if 'FP' is used, 'prune_interval' must be set.";
                FilterPrune();
            } else if (DeepCompression::prune_method == "PP") {
                ProbPrune();
            } /// TODO: change to switch
        }   
    } else {
        if (DeepCompression::prune_method == "PP") {
            Dtype rands[num_col];
            caffe_rng_uniform(num_col, (Dtype)0, (Dtype)1, rands);
            for (int i = 0; i < count; ++i) {
                this->masks_[i] = rands[i % num_col] < DeepCompression::history_prob[this->layer_index][i % num_col] ? 1 : 0; /// geerate masks
            }              
            for (int i = 0; i < count; ++i) { this->weight_backup[i] = muweight[i]; } /// backup weights
            this->IF_restore = true;
            for (int i = 0; i < count; ++i) { muweight[i] *= this->masks_[i]; } /// do pruning
        }
    }
  /// ------------------------------------------------------
    const Dtype* weight = this->blobs_[0]->gpu_data();
    for (int i = 0; i < bottom.size(); ++i) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        Dtype* top_data = top[i]->mutable_gpu_data();
        for (int n = 0; n < this->num_; ++n) {
            this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
            top_data + n * this->top_dim_);
            if (this->bias_term_) {
                const Dtype* bias = this->blobs_[1]->gpu_data();
                this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
            }
        }
    }
    
    /// Print feature map to check --------
    /// If row 3 and 8 are pruned in previous layer, then channel 3 and 8 will be only biases in this layer's feature map.
    /**
    if (!DeepCompression::IN_TEST && this->layer_index == 0) {
        cout << "bottom.size(): " << bottom.size() << endl;
        for (int i = 0; i < bottom.size(); ++i) {
            const Dtype* top_data = top[i]->cpu_data();
            const int channel = top[i]->shape()[1];
            const int width   = top[i]->shape()[2];
            const int height  = top[i]->shape()[3];
            cout << "channel: " << channel << " " << width << " " <<  height << endl;
            
            vector<Dtype> sum(channel, 0);
            for (int c = 0; c < channel; ++c) {
                for (int w = 0 ; w < width; ++w) {
                    for (int h = 0; h < height; ++h) {
                        sum[c] += fabs(top_data[0 + c * width * height + w * height + h]);
                    }
                }
            }
            for (int c = 0; c < channel; ++c) {
                cout << sum[c] << "  ";
            }
            cout << endl;
        }
    }
    */
    /// -----------------------------------
    
    
    
    /// Restore weights ----------------
    if (this->IF_restore) {
        /// cout << layer_name << ": restore weights! " << endl;
        this->blobs_[0]->mutable_cpu_data();
        /// this->blobs_[0]->gpu_data(); 
        /// Interesting! If the above line is added, something like "control" seems to transfer from cpu to gpu. 
        /// Then modifying cpu weights won't affect their gpu counterparts.
        for (int i = 0; i < count; ++i) {
            muweight[i] = this->weight_backup[i];
        }
        
        /**
        /// ========================
        /// Chech restore
        cout << "weights from cpu:" << endl;
        for (int i = 0; i < 20; ++i) {
            cout << muweight[i] << " ";
        }
        cout << endl;

        Dtype weight_cpu[count];
        const Dtype* weight_gpu = this->blobs_[0]->gpu_data();
        cout << "weights copied from gpu:" << endl;
        hipMemcpy(weight_cpu, weight_gpu, sizeof(Dtype) * count, hipMemcpyDeviceToHost);
        for (int i = 0; i < 20; ++i) {
            cout << weight_cpu[i] << " ";
        }
        cout << endl;
        /// ========================
        */
    }
    /// --------------------------------
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff);
        }
        
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_);
        }
      }
    }
  }
  
  /// ADDED BY WANGHUAN ------------------------------------------
  Dtype* muweight_diff = this->blobs_[0]->mutable_cpu_diff();      
  const int count = this->blobs_[0]->count();
  
  /// UpdateDiffs(); /// update second diff and so on
  
  /// Print and check
  const string layer_name = this->layer_param_.name();
  if (layer_name == "conv2" && DeepCompression::step_ % SHOW_INTERVAL == 0) {
      cout.width(5);  cout << "Index" << "   ";
      cout.width(16); cout << "DiffBeforeMasked" << "   ";
      cout.width(4);  cout << "Mask" << "   ";
      cout.width(4);  cout << "Prob" << endl;
      for (int i = 0; i < 20; ++i) {
          cout.width(3);  cout << "#";
          cout.width(2);  cout << i+1 << "   ";
          cout.width(16); cout << muweight_diff[i] << "   ";
          cout.width(4);  cout << this->masks_[i] << "   ";
          cout.width(4);  cout << DeepCompression::history_prob[this->layer_index][i] << endl;
      }
  }
  
  /// Apply masks to diff
  const bool IF_mask = DeepCompression::IN_RETRAIN || (DeepCompression::step_ - 1) >= DeepCompression::prune_begin_iter;
  if (IF_mask) {
      if (DeepCompression::prune_method == "Prune" && DeepCompression::criteria == "diff") { } /// UpdateMasks(); }
      for (int j = 0; j < count; ++j) { muweight_diff[j] *= this->masks_[j]; }
  }
  /// ------------------------------------------------------------- 
  
  
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayer);

}  // namespace caffe
