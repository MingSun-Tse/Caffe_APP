#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/layers/conv_layer.hpp"
#include "caffe/adaptive_probabilistic_pruning.hpp"

using namespace std;
namespace caffe {

template <typename Dtype>
void ConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
    this->PruneForward(); /// @mingsuntse, for pruning
    const Dtype* weight = this->blobs_[0]->gpu_data();
    for (int i = 0; i < bottom.size(); ++i) {
        const Dtype* bottom_data    = bottom[i]->gpu_data();
        Dtype* top_data    = top[i]->mutable_gpu_data();
        for (int n = 0; n < this->num_; ++n) {
            this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight, top_data + n * this->top_dim_);
            if (this->bias_term_) {
                const Dtype* bias = this->blobs_[1]->gpu_data();
                this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
            }
        }
    }
    /// Print feature map to check --------
    /// If row 3 and 8 are pruned in previous layer, then channel 3 and 8 will be only biases in this layer's feature map.
    /**
    if (!APP<Dtype>::IN_TEST && L == 0) {
        cout << "bottom.size(): " << bottom.size() << endl;
        for (int i = 0; i < bottom.size(); ++i) {
            const Dtype* top_data = top[i]->cpu_data();
            const int channel = top[i]->shape()[1];
            const int width   = top[i]->shape()[2];
            const int height  = top[i]->shape()[3];
            cout << "channel: " << channel << " " << width << " " <<  height << endl;
            
            vector<Dtype> sum(channel, 0);
            for (int c = 0; c < channel; ++c) {
                for (int w = 0 ; w < width; ++w) {
                    for (int h = 0; h < height; ++h) {
                        sum[c] += fabs(top_data[0 + c * width * height + w * height + h]);
                    }
                }
            }
            for (int c = 0; c < channel; ++c) {
                cout << sum[c] << "  ";
            }
            cout << endl;
        }
    }
    */
    
    // Restore weights when using ProbPrune
    if (this->IF_restore) {
        /*
        /// cout << layer_name << ": restore weights! " << endl;
        Dtype* muweight = this->blobs_[0]->mutable_cpu_data();
        /// this->blobs_[0]->gpu_data(); 
        /// Interesting! If the above line is added, something like "control" seems to transfer from cpu to gpu. 
        /// Then modifying cpu weights won't affect their gpu counterparts.
        for (int i = 0; i < count; ++i) {
            muweight[i] = this->weight_backup[i];
        }
        */
        caffe_gpu_memcpy(this->blobs_[0]->count(),
                         this->blobs_backup_[0]->gpu_data(),
                         this->blobs_[0]->mutable_gpu_data());
    }
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  
  caffe_gpu_mul(this->blobs_[0]->count(),
                this->blobs_[0]->gpu_data(),
                this->blobs_[0]->gpu_data(),
                this->blobs_[0]->mutable_gpu_secdata()); // w^2
  
  const Dtype* weight    = this->blobs_[0]->gpu_data();
  const Dtype* secweight = this->blobs_[0]->gpu_secdata();
  Dtype* weight_diff    = this->blobs_[0]->mutable_gpu_diff();
  Dtype* weight_secdiff = this->blobs_[0]->mutable_gpu_secdiff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff    = top[i]->gpu_diff();
    const Dtype* top_secdiff = top[i]->gpu_secdiff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff    = this->blobs_[1]->mutable_gpu_diff();
      Dtype* bias_secdiff = this->blobs_[1]->mutable_gpu_secdiff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff,    top_diff    + n * this->top_dim_);
        this->backward_gpu_bias(bias_secdiff, top_secdiff + n * this->top_dim_); // TODO(mingsuntse): check this, maybe wrong.
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      
      caffe_gpu_mul(bottom[i]->count(),
                    bottom_data,
                    bottom_data,
                    bottom[i]->mutable_gpu_secdata()); // x^2
      const Dtype* bottom_secdata = bottom[i]->gpu_secdata();
      
      Dtype* bottom_diff    = bottom[i]->mutable_gpu_diff();
      Dtype* bottom_secdiff = bottom[i]->mutable_gpu_secdiff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data    + n * this->bottom_dim_, top_diff    + n * this->top_dim_, weight_diff);
          this->weight_gpu_gemm(bottom_secdata + n * this->bottom_dim_, top_secdiff + n * this->top_dim_, weight_secdiff); /// Added by @mingsuntse
        }
        
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff    + n * this->top_dim_, weight,    bottom_diff    + n * this->bottom_dim_);
          this->backward_gpu_gemm(top_secdiff + n * this->top_dim_, secweight, bottom_secdiff + n * this->bottom_dim_); /// Added by @mingsuntse
        } 
      }
    }
  }
  this->PruneBackward(top); /// @mingsuntse, for pruning
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayer);

}  // namespace caffe
