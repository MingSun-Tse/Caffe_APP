#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/layers/conv_layer.hpp"
#include "caffe/adaptive_probabilistic_pruning.hpp"
#define SHOW_INTERVAL 10
#define SHOW_NUM_LAYER 5

using namespace std;
namespace caffe {

template <typename Dtype>
void ConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
          
    /// ADDED BY WANGHUAN -----------------------------------
    Dtype* muweight = this->blobs_[0]->mutable_cpu_data();
    const int count = this->blobs_[0]->count();
    const int num_row = this->blobs_[0]->shape()[0];
    const int num_col = count / num_row;
    const string layer_name = this->layer_param_.name();
    const string mthd = APP::prune_method;
    const int L = APP::layer_index[layer_name];
    this->IF_restore = false;
    
    
    /// IF_mask
    const bool IF_prune       = mthd != "None";
    const bool IF_enough_iter = (APP::step_ - 1) >= APP::prune_begin_iter;
    const bool IF_pruned      = APP::pruned_ratio[L] > 0;
    this->IF_mask             = IF_prune && (IF_enough_iter || IF_pruned);
    
    
    if (this->phase_ == TRAIN) {
        if (this->IF_mask) {
            
            // UpdateNumPrunedRow/Col
            // Note that, UpdateNumPrunedRow/Col before pruning, 
            // so that when calculating score, the zombie weights will not be counted.
            if ((mthd == "PPc" || mthd == "CP") && L != APP::layer_cnt-1) {
                if (APP::step_ - 1 - APP::iter_prune_finished[L + 1] <= 1) {
                    UpdateNumPrunedRow();
                }
            } else if ((mthd == "PPr" || mthd == "FP" || mthd == "TP") && L != 0) {
                UpdateNumPrunedCol();
            }
            UpdatePrunedRatio();
            
            // Given pruned_ratio, judge whether prune finished for current layer
            // Get into here ONLY once
            if (APP::iter_prune_finished[L] == INT_MAX) {
                const Dtype pruned_ratio = (mthd == "PPr" || mthd == "FP" || mthd == "TP") 
                                ? APP::pruned_ratio_row[L] : APP::pruned_ratio_col[L];
                const bool layer_finish = (pruned_ratio >= APP::prune_ratio[L]); /// layer pruning target achieved
                const bool net_finish   = APP::IF_speedup_achieved; /// net pruning target achieved
                if (layer_finish || net_finish) {
                    if (mthd.substr(0, 2) == "PP") { CleanWorkForPP(); } /// last time, do some clean work
                    APP::iter_prune_finished[L] = APP::step_ - 1;
                    cout << layer_name << " prune finished!" 
                         << "  step: " << APP::step_ 
                         << "  pruned_ratio: "     << APP::pruned_ratio[L]
                         << "  pruned_ratio_col: " << APP::pruned_ratio_col[L] << endl;
                    IF_alpf();
                }
            }
        }
        
        // Print and check
        if (mthd != "None" && L < SHOW_NUM_LAYER && APP::inner_iter == 0) {
            cout << layer_name << "  IF_mask: " << this->IF_mask 
                 << "  pruned_ratio: " << APP::pruned_ratio[L]
                 << "  prune_ratio: "  << APP::prune_ratio[L]
                 << "  num_pruned_col: " << APP::num_pruned_col[L]
                 << "  num_pruned_row: " << APP::num_pruned_row[L] << endl;
        }
        if (L == 1 && APP::step_ % SHOW_INTERVAL == 0 && APP::inner_iter == 0) {
            Print(L, 'f');
        }

        // Update masks and apply masks
        if (this->IF_mask && APP::iter_prune_finished[L] == INT_MAX) {
            if (mthd == "Prune" && APP::criteria == "L2-norm") { 
                /// UpdateMasks(); 
            } else if (mthd == "FP" && (APP::step_ - 1) % APP::prune_interval == 0) {
                FilterPrune(); 
            } else if (mthd == "PPc" && IF_hppf()) {
                ProbPruneCol();
            } else if (mthd == "PPr" && IF_hppf()) {
                ProbPruneRow();
            }  else if (mthd == "TP") {
                for (int i = 0; i < count; ++i) {
                    muweight[i] *= APP::masks[L][i]; /// explictly prune, because seems TP is wrong somewhere.
                }  
            }
        }
        
        // Weight logging
        if (APP::num_log) {
            const int num_log = APP::log_index[L].size();
            for (int k = 0; k < num_log; ++k) {
                const int index = APP::log_index[L][k];
                Dtype sum = 0;
                for (int i = 0; i < num_row; ++i) {
                    sum += fabs(muweight[i * num_col + index]);
                }
                sum /= num_row;
                APP::log_weight[L][k].push_back(sum);
            }
        }
    } else {
        if (this->IF_mask && APP::iter_prune_finished[L] == INT_MAX && mthd == "PP") {
            Dtype rands[num_col];
            caffe_rng_uniform(num_col, (Dtype)0, (Dtype)1, rands);
            for (int i = 0; i < count; ++i) {
                APP::masks[L][i] = rands[i % num_col] < APP::history_prob[L][i % num_col] ? 1 : 0; /// generate masks
            }              
            for (int i = 0; i < count; ++i) { 
                this->weight_backup[i] = muweight[i]; /// backup weights
            } 
            this->IF_restore = true;
            for (int i = 0; i < count; ++i) { 
                muweight[i] *= APP::masks[L][i]; /// apply masks
            } 
        }
    }
  /// ------------------------------------------------------
  
    const Dtype* weight = this->blobs_[0]->gpu_data();
    for (int i = 0; i < bottom.size(); ++i) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        Dtype* top_data = top[i]->mutable_gpu_data();
        for (int n = 0; n < this->num_; ++n) {
            this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
            top_data + n * this->top_dim_);
            if (this->bias_term_) {
                const Dtype* bias = this->blobs_[1]->gpu_data();
                this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
            }
        }
    }
    /// this->bottom_dim_: bottom feature map size, input
    /// this->top_dim_: top feature map size, output
    /// this->num_: batch size
    
    /// Print feature map to check --------
    /// If row 3 and 8 are pruned in previous layer, then channel 3 and 8 will be only biases in this layer's feature map.
    /**
    if (!APP::IN_TEST && L == 0) {
        cout << "bottom.size(): " << bottom.size() << endl;
        for (int i = 0; i < bottom.size(); ++i) {
            const Dtype* top_data = top[i]->cpu_data();
            const int channel = top[i]->shape()[1];
            const int width   = top[i]->shape()[2];
            const int height  = top[i]->shape()[3];
            cout << "channel: " << channel << " " << width << " " <<  height << endl;
            
            vector<Dtype> sum(channel, 0);
            for (int c = 0; c < channel; ++c) {
                for (int w = 0 ; w < width; ++w) {
                    for (int h = 0; h < height; ++h) {
                        sum[c] += fabs(top_data[0 + c * width * height + w * height + h]);
                    }
                }
            }
            for (int c = 0; c < channel; ++c) {
                cout << sum[c] << "  ";
            }
            cout << endl;
        }
    }
    */
    /// -----------------------------------
    
    
    
    /// Restore weights ----------------
    if (this->IF_restore) {
        /// cout << layer_name << ": restore weights! " << endl;
        this->blobs_[0]->mutable_cpu_data();
        /// this->blobs_[0]->gpu_data(); 
        /// Interesting! If the above line is added, something like "control" seems to transfer from cpu to gpu. 
        /// Then modifying cpu weights won't affect their gpu counterparts.
        for (int i = 0; i < count; ++i) {
            muweight[i] = this->weight_backup[i];
        }
        
        /**
        /// ========================
        /// Chech restore
        cout << "weights from cpu:" << endl;
        for (int i = 0; i < 20; ++i) {
            cout << muweight[i] << " ";
        }
        cout << endl;

        Dtype weight_cpu[count];
        const Dtype* weight_gpu = this->blobs_[0]->gpu_data();
        cout << "weights copied from gpu:" << endl;
        hipMemcpy(weight_cpu, weight_gpu, sizeof(Dtype) * count, hipMemcpyDeviceToHost);
        for (int i = 0; i < 20; ++i) {
            cout << weight_cpu[i] << " ";
        }
        cout << endl;
        /// ========================
        */
    }
    /// --------------------------------
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff);
        }
        
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_);
        }
      }
    }
  }
  
/// ADDED BY WANGHUAN ------------------------------------------
    Dtype* muweight_diff = this->blobs_[0]->mutable_cpu_diff();      
    const int count   = this->blobs_[0]->count();
    const int num_row = this->blobs_[0]->shape()[0];
    const int num_col = count / num_row;
    const int L = APP::layer_index[this->layer_param_.name()];

    /// Print and check
    if (L == 1 && APP::step_ % SHOW_INTERVAL == 0 && APP::inner_iter == 0) {
        Print(L, 'b');
    }
    
    /// Diff log
    if (APP::num_log) {
        const int num_log = APP::log_index[L].size();
        for (int i = 0; i < num_log; ++i) {
            const int index = APP::log_index[L][i];
            Dtype sum = 0;
            for (int r = 0; r < num_row; ++r) {
                sum += fabs(muweight_diff[r * num_col + index]);
            }
            sum /= num_row;
            APP::log_diff[L][i].push_back(sum);
        }
    }
    
    if (this->IF_mask) {
        for (int j = 0; j < count; ++j) { 
            muweight_diff[j] *= APP::masks[L][j]; 
        }
        if (APP::iter_prune_finished[L] == INT_MAX) {
            if (APP::prune_method == "Prune" && APP::criteria == "diff") {
            } else if (APP::prune_method == "TP" && (APP::step_ - 1) % APP::prune_interval == 0) {
                TaylorPrune(top);
            }
        }
    }
/// ------------------------------------------------------------- 
  
  
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayer);

}  // namespace caffe
