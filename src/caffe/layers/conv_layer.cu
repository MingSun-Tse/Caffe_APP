#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/layers/conv_layer.hpp"
#include "caffe/adaptive_probabilistic_pruning.hpp"
#define SHOW_INTERVAL 10

using namespace std;
namespace caffe {

template <typename Dtype>
void ConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
          
    /// ADDED BY WANGHUAN -----------------------------------
    Dtype* muweight = this->blobs_[0]->mutable_cpu_data();
    const int count = this->blobs_[0]->count();
    const int num_row = this->blobs_[0]->shape()[0];
    const int num_col = count / num_row;
    const string layer_name = this->layer_param_.name();
    this->IF_restore = false;
    
    /// IF_mask
    const bool IF_prune       = APP::prune_method != "None";
    const bool IF_enough_iter = (APP::step_ - 1) >= APP::prune_begin_iter;
    const bool IF_pruned      = this->pruned_ratio > 0;
    const bool IF_mask        = IF_prune && (IF_enough_iter || IF_pruned);
    
    if (this->phase_ == TRAIN) {
        if (IF_mask) {
            // UpdateNumPrunedRow();
            // UpdateNumPrunedCol();
            this->pruned_ratio = 1 - (1 - this->num_pruned_col * 1.0 / num_col) * (1 - this->num_pruned_row * 1.0 / num_row);
            if (!APP::IF_prune_finished[this->layer_index]) {
                if (this->pruned_ratio >= this->prune_ratio) {
                    if (APP::prune_method == "PP") { CleanWorkForPP(); } // last time, do some clean work
                    APP::IF_prune_finished[this->layer_index] = true;
                    cout << layer_name << " prune finished!" 
                         << "  step: " << APP::step_ 
                         << "  pruned_ratio: " << this->pruned_ratio << endl;
                }
            }
        }
        
        /// Print and check
        if (APP::prune_method != "None" && this->layer_index < 5 && APP::inner_iter == 0) {
            cout << layer_name << "  IF_mask: " << IF_mask 
                 << "  pruned_ratio: " << this->pruned_ratio
                 << "  prune_ratio: " << this->prune_ratio 
                 << "  num_pruned_col: " << this->num_pruned_col
                 << "  num_pruned_row: " << this->num_pruned_row << endl;
        }
        
        /// Print and check (before pruning)
        if (this->layer_index == 1 && APP::step_ % SHOW_INTERVAL == 0 && APP::inner_iter == 0) {
            /// cout.setf(std::ios::left);
            cout.width(5);  cout << "Index" << "   ";
            cout.width(18); cout << "WeightBeforeMasked" << "   ";
            cout.width(4);  cout << "Mask" << "   ";
            cout.width(4);  cout << "Prob" << endl;
            for (int i = 0; i < 20; ++i) {
                cout.width(3);  cout << "#";
                cout.width(2);  cout << i+1 << "   ";
                cout.width(18); cout << muweight[i] << "   ";
                cout.width(4);  cout << this->masks_[i] << "   ";
                cout.width(4);  cout << APP::history_prob[this->layer_index][i] << endl;
            }
        }

        /// Update masks and apply masks
        if (IF_mask && this->pruned_ratio < this->prune_ratio) {
            if (APP::prune_method == "Prune" && APP::criteria == "L2-norm") { 
                /// UpdateMasks(); 
            } else if (APP::prune_method == "FP") {
                CHECK_GE(APP::prune_interval, 1)
                        << "Error: if 'FP' is used, 'prune_interval' must be set.";
                if ((APP::step_ - 1) % APP::prune_interval == 0) { FilterPrune(); }    
            } else if (APP::prune_method == "PP") {
                bool IF_hppf = true; /// IF_higher_priority_prune_finished 
                for (int i = 0; i <= APP::layer_cnt[0]; ++i) {
                    if (APP::priority[i] < APP::priority[this->layer_index] && !APP::IF_prune_finished[i]) {
                        IF_hppf = false;
                        break;
                    }
                }
                if (IF_hppf) { ProbPrune(); }
            }  else if (APP::prune_method == "TP") {
                for (int i = 0; i < count; ++i) {
                    muweight[i] *= this->masks_[i]; 
                }  // explictly prune, because seems TP is wrong somewhere.
            }
            
        }  
        bool IF_log = true;
        if (IF_log) {
            const int num_log = APP::log_index[this->layer_index].size();
            for (int i = 0; i < num_log; ++i) {
                const int index = APP::log_index[this->layer_index][i];
                Dtype sum = 0;
                for (int i = 0; i < num_row; ++i) {
                    sum += fabs(muweight[i * num_col + index]);
                }
                sum /= num_row;
                APP::log_weight[this->layer_index][i].push_back(sum);
            }
        }
    } else {
        if (APP::prune_method == "PP") {
            Dtype rands[num_col];
            caffe_rng_uniform(num_col, (Dtype)0, (Dtype)1, rands);
            for (int i = 0; i < count; ++i) {
                this->masks_[i] = rands[i % num_col] < APP::history_prob[this->layer_index][i % num_col] ? 1 : 0; /// geerate masks
            }              
            for (int i = 0; i < count; ++i) { this->weight_backup[i] = muweight[i]; } /// backup weights
            this->IF_restore = true;
            for (int i = 0; i < count; ++i) { muweight[i] *= this->masks_[i]; } /// do pruning
        }
    }
    

    
  /// ------------------------------------------------------
    const Dtype* weight = this->blobs_[0]->gpu_data();
    for (int i = 0; i < bottom.size(); ++i) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        Dtype* top_data = top[i]->mutable_gpu_data();
        for (int n = 0; n < this->num_; ++n) {
            this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
            top_data + n * this->top_dim_);
            if (this->bias_term_) {
                const Dtype* bias = this->blobs_[1]->gpu_data();
                this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
            }
        }
    }
    /// this->bottom_dim_: bottom feature map size, input
    /// this->top_dim_: top feature map size, output
    /// this->num_: batch size
    
    /// Print feature map to check --------
    /// If row 3 and 8 are pruned in previous layer, then channel 3 and 8 will be only biases in this layer's feature map.
    /**
    if (!APP::IN_TEST && this->layer_index == 0) {
        cout << "bottom.size(): " << bottom.size() << endl;
        for (int i = 0; i < bottom.size(); ++i) {
            const Dtype* top_data = top[i]->cpu_data();
            const int channel = top[i]->shape()[1];
            const int width   = top[i]->shape()[2];
            const int height  = top[i]->shape()[3];
            cout << "channel: " << channel << " " << width << " " <<  height << endl;
            
            vector<Dtype> sum(channel, 0);
            for (int c = 0; c < channel; ++c) {
                for (int w = 0 ; w < width; ++w) {
                    for (int h = 0; h < height; ++h) {
                        sum[c] += fabs(top_data[0 + c * width * height + w * height + h]);
                    }
                }
            }
            for (int c = 0; c < channel; ++c) {
                cout << sum[c] << "  ";
            }
            cout << endl;
        }
    }
    */
    /// -----------------------------------
    
    
    
    /// Restore weights ----------------
    if (this->IF_restore) {
        /// cout << layer_name << ": restore weights! " << endl;
        this->blobs_[0]->mutable_cpu_data();
        /// this->blobs_[0]->gpu_data(); 
        /// Interesting! If the above line is added, something like "control" seems to transfer from cpu to gpu. 
        /// Then modifying cpu weights won't affect their gpu counterparts.
        for (int i = 0; i < count; ++i) {
            muweight[i] = this->weight_backup[i];
        }
        
        /**
        /// ========================
        /// Chech restore
        cout << "weights from cpu:" << endl;
        for (int i = 0; i < 20; ++i) {
            cout << muweight[i] << " ";
        }
        cout << endl;

        Dtype weight_cpu[count];
        const Dtype* weight_gpu = this->blobs_[0]->gpu_data();
        cout << "weights copied from gpu:" << endl;
        hipMemcpy(weight_cpu, weight_gpu, sizeof(Dtype) * count, hipMemcpyDeviceToHost);
        for (int i = 0; i < 20; ++i) {
            cout << weight_cpu[i] << " ";
        }
        cout << endl;
        /// ========================
        */
    }
    /// --------------------------------
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff);
        }
        
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_);
        }
      }
    }
  }
  
/// ADDED BY WANGHUAN ------------------------------------------
    Dtype* muweight_diff = this->blobs_[0]->mutable_cpu_diff();      
    const int count = this->blobs_[0]->count();
    const int num_row = this->blobs_[0]->shape()[0];
    const int num_col = count / num_row;
    
    /// UpdateDiffs(); /// update second diff and so on

    /// Print and check
    if (this->layer_index == 1 && APP::step_ % SHOW_INTERVAL == 0 && APP::inner_iter == 0) {
        cout.width(5);  cout << "Index" << "   ";
        cout.width(16); cout << "DiffBeforeMasked" << "   ";
        cout.width(4);  cout << "Mask" << "   ";
        cout.width(4);  cout << "Prob" << endl;
        for (int i = 0; i < 20; ++i) {
            cout.width(3);  cout << "#";
            cout.width(2);  cout << i+1 << "   ";
            cout.width(16); cout << muweight_diff[i] << "   ";
            cout.width(4);  cout << this->masks_[i] << "   ";
            cout.width(4);  cout << APP::history_prob[this->layer_index][i] << endl;
        }
    }
    
    /// Diff log
    bool IF_log = true;
    if (IF_log) {
        const int num_log = APP::log_index[this->layer_index].size();
        for (int i = 0; i < num_log; ++i) {
            const int index = APP::log_index[this->layer_index][i];
            Dtype sum = 0;
            for (int r = 0; r < num_row; ++r) {
                sum += fabs(muweight_diff[r * num_col + index]);
            }
            sum /= num_row;
            APP::log_diff[this->layer_index][i].push_back(sum);
        }
    }
    

    /// IF_mask
    const bool IF_prune       = APP::prune_method != "None";
    const bool IF_enough_iter = (APP::step_ - 1) >= APP::prune_begin_iter;
    const bool IF_pruned      = this->pruned_ratio > 0;
    const bool IF_mask        = IF_prune && (IF_enough_iter || IF_pruned) ;
    if (IF_mask) {
        for (int j = 0; j < count; ++j) { muweight_diff[j] *= this->masks_[j]; }
        if (this->pruned_ratio < this->prune_ratio) {
            if (APP::prune_method == "Prune" && APP::criteria == "diff") {
                /// UpdateMasks(); 
            } else if (APP::prune_method == "TP") {
                CHECK_GE(APP::prune_interval, 1)
                    << "Error: if 'TP' is used, 'prune_interval' must be set.";
                if ((APP::step_ - 1) % APP::prune_interval == 0) { TaylorPrune(top); }
            }
        }
    }
    


/// ------------------------------------------------------------- 
  
  
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayer);

}  // namespace caffe
