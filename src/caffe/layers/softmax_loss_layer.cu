#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/softmax_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SoftmaxLossForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* label, Dtype* loss,
          const int num, const int dim, const int spatial_dim,
          const bool has_ignore_label_, const int ignore_label_,
          Dtype* counts) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);
    if (has_ignore_label_ && label_value == ignore_label_) {
      loss[index] = 0;
      counts[index] = 0;
    } else {
      loss[index] = -log(max(prob_data[n * dim + label_value * spatial_dim + s],
                      Dtype(FLT_MIN)));
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
void SoftmaxWithLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_); // Calculate prob_
  const Dtype* prob_data = prob_.gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const int dim = prob_.count() / outer_num_;
  const int nthreads = outer_num_ * inner_num_;
  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  // Similarly, this memory is never used elsewhere, and thus we can use it
  // to avoid having to allocate additional GPU memory.
  Dtype* counts = prob_.mutable_gpu_diff();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SoftmaxLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, label, loss_data,
      outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts); // Calculate cross entropy loss
  Dtype loss;
  caffe_gpu_asum(nthreads, loss_data, &loss); // Get total loss
  Dtype valid_count = -1;
  // Only launch another CUDA kernel if we actually need the count of valid
  // outputs.
  if (normalization_ == LossParameter_NormalizationMode_VALID &&
      has_ignore_label_) {
    caffe_gpu_asum(nthreads, counts, &valid_count);
  }
  top[0]->mutable_cpu_data()[0] = loss / get_normalizer(normalization_,
                                                        valid_count);
  if (top.size() == 2) {
    top[1]->ShareData(prob_);
  }
}

template <typename Dtype>
__global__ void SoftmaxLossBackwardGPU(const int nthreads, const Dtype* top,
          const Dtype* label, Dtype* bottom_diff, const int num, const int dim,
          const int spatial_dim, const bool has_ignore_label_,
          const int ignore_label_, Dtype* counts) {
  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) { /// @mingsuntse, nthreads = batch_size, here 
    const int n = index / spatial_dim; // n = {0, 1, ..., batch_size - 1}
    const int s = index % spatial_dim; // s = 0 here
    const int label_value = static_cast<int>(label[n * spatial_dim + s]); // n * spatial_dim + s = index, why do not use index directly?
    if (has_ignore_label_ && label_value == ignore_label_) {
      for (int c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] = 0;
      }
      counts[index] = 0;
    } else {
      bottom_diff[n * dim + label_value * spatial_dim + s] -= 1;
      counts[index] = 1;
      
    }
  }
}

template <typename Dtype>
void SoftmaxWithLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff(); // bottom[0] is logits, bottom[1] is labels
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* top_data = top[0]->gpu_data();
    caffe_gpu_memcpy(prob_.count() * sizeof(Dtype), prob_data, bottom_diff); // Initialize to p, note that, dLoss/dzi = pi - qi
    
    /*
    // second diff of logits = pi + pi * pi
    caffe_gpu_mul(prob_.count(),
                  prob_data,
                  prob_data,
                  bottom[0]->mutable_gpu_secdiff());
    caffe_gpu_add(prob_.count(),
                  prob_data,
                  bottom[0]->gpu_secdiff(),
                  bottom[0]->mutable_gpu_secdiff()); /// @mingsuntse
    */              
    const Dtype* label = bottom[1]->gpu_data();
    const int dim = prob_.count() / outer_num_;
    const int nthreads = outer_num_ * inner_num_;
    // Since this memory is never used for anything else,
    // we use to to avoid allocating new GPU memory.
    Dtype* counts = prob_.mutable_gpu_diff();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SoftmaxLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, top_data, label, bottom_diff,
        outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts); // get bottom_diff
        
    /* Check
    const Dtype* label_cpu = bottom[1]->cpu_data();
    std::cout << nthreads
          << " " << prob_.count()      // = batch_size * num_class, bottom[0] has the same dimension
          << " " << bottom[1]->count() // = batch_size, because bottom[1] is labels, each exampple has one label, whose range = {0, num_class - 1}
          << " " << outer_num_         // = batch_size
          << " " << dim                // = num_class
          << " " << inner_num_         // spatial_dim, = 1 here. Any occasions that inner_num_ != 1 ?
          << std::endl;
    for (int i = 0; i < bottom[1]->count(); ++i) {
        std::cout << label_cpu[i] << " ";
    }
    std::cout << std::endl;
    */
        
    Dtype valid_count = -1;
    // Only launch another CUDA kernel if we actually need the count of valid
    // outputs.
    if (normalization_ == LossParameter_NormalizationMode_VALID &&
        has_ignore_label_) {
      caffe_gpu_asum(nthreads, counts, &valid_count);
    }
    const Dtype loss_weight = top[0]->cpu_diff()[0] / get_normalizer(normalization_, valid_count); // normalizer = bacth_size for convnet exp
    // std::cout << "valid_count: " << valid_count // -1 for convnet exp
               // << "  loss_weight: " << loss_weight << std::endl; 
    // loss_weight is the multiplier for a loss term in total loss, usually used in the multi-loss task
    caffe_gpu_scal(prob_.count(), loss_weight , bottom_diff);
    // caffe_gpu_scal(prob_.count(), loss_weight , bottom[0]->mutable_gpu_secdiff()); /// @mingsuntse
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxWithLossLayer);

}  // namespace caffe
